#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <time.h>
#include <winsock2.h>
#include <malloc.h>
#include <chrono>
#include <Windows.h>
#include <stdint.h>
#include <iostream>

// Функция для сложения векторов на CPU
void addVectorCPU(long long* a, long long* b, long long* c, long long n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

__global__ void vectorAdd(long long* a, long long* b, long long* c, long long n)
{
        
    int i = blockIdx.x * blockDim.x + threadIdx.x; // номер блока, кол-во потоков в блоке, индекс тек потока
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

void print(long long* c, long long N)
{
    std::cout << "first 50 el\n\n\n";

    for (int i = 0; i < 50; i++)printf("%d ", c[i]);

    std::cout << "\n\n";

    std::cout << "\n\nLast 50 el\n\n";

    for (int i = N - 50; i < N; i++)printf("%d ", c[i]);

    std::cout << "\n\n";
}

int main() 
{
    const long long N = 100000000;  // Длина векторов
    long long* a, * b, * c;       // Входные и выходной векторы на CPU
    long long* d_a, * d_b, * d_c; // Входные и выходной векторы на GPU


    a = (long long*)malloc(N * sizeof(long long));
    b = (long long*)malloc(N * sizeof(long long));
    c = (long long*)malloc(N * sizeof(long long));

    hipMalloc((void**)&d_a, N * sizeof(long long));
    hipMalloc((void**)&d_b, N * sizeof(long long));
    hipMalloc((void**)&d_c, N * sizeof(long long));

    // Заполнение векторов значениями
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i;
    }

    hipMemcpy(d_a, a, N * sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(long long), hipMemcpyHostToDevice);

    // Запуск ядра функции vectorAdd на GPU и замерка времени выполнения
    
    auto start = std::chrono::high_resolution_clock::now();


    const int block_size = 1024;
    int num_blocks = (N + block_size - 1) / block_size; 
    vectorAdd <<< num_blocks, block_size>> > (d_a, d_b, d_c, N);

    //vectorAdd <<< 1024, 1024 >>> (d_a, d_b, d_c, N);
    hipDeviceSynchronize();    // Ожидание завершения всех операций на устройстве
    
    
    auto end = std::chrono::high_resolution_clock::now();
    
    std::chrono::duration<double> duration = end - start;

    hipMemcpy(c, d_c, N * sizeof(long long), hipMemcpyDeviceToHost);
    
    

    print(c, N);
    printf("\nGPU Time for N=%d: %lf ms\n", N, duration.count());



    //Выполнение операции сложения на CPU и замерка времени
    start = std::chrono::high_resolution_clock::now();


    addVectorCPU(a, b, c, N);


    end = std::chrono::high_resolution_clock::now();

   duration = end - start;

   print(c, N);

    printf("CPU Time for N=%d: %f ms\n", N, duration);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}


//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//
//
//
//
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    static int z;
//    z++;
//    int i = threadIdx.x;
//    c[z] = a[i] + b[i];
//}
//
//
//__global__ void vectorAdd(int* a, int* b, int* c, int length) {
//    int tid = threadIdx.x + blockIdx.x * blockDim.x;
//    if (tid < length) {
//        c[tid] = a[tid] + b[tid];
//    }
//}
//
//void sumCPU(int sum, int size) {
//    for (int i = 0; i < size; i++)sum++;
//}
//
//int main()
//{
//    int sumCPU = 0, sumGPU = 0;
//
//       
//
//
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//
//
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//    
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
